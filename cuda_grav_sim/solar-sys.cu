#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <unordered_map>
#include <cmath>
#include <string>
#include <fstream>
#include <sstream>
#include <random>
#include <chrono>
#include <unistd.h>

// There is no c++20 compatibility for the nvcc on amarel
// #include <numbers> 

using namespace std;

#define PI 3.14159265358979323846f

// Constants
const double year = 365.25 * 24 * 60 * 60;
const float G = 6.67e-11f;
random_device rd;
mt19937 gen(0); // Seeded for reproducibility
uniform_real_distribution<> dis(0, 1);

const int print_every = 100;
const int graph_every = 1000;

struct body {
    uint32_t id;
    float Gm;
    float x, y, z;
    float vx, vy, vz;
    float ax, ay, az;
    float old_ax, old_ay, old_az;
};

struct Bodies {
    float *Gm;
    float *x, *y, *z;
    float *vx, *vy, *vz;
    float *ax, *ay, *az;
    float *old_ax, *old_ay, *old_az;
};

// CUDA Kernels

__global__ void compute_acceleration_kernel(int n, Bodies bodies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float ax = 0.0f;
    float ay = 0.0f;
    float az = 0.0f;

    float x1 = bodies.x[i];
    float y1 = bodies.y[i];
    float z1 = bodies.z[i];

    for (int j = 0; j < n; j++) {
        if (i == j) continue;

        float dx = bodies.x[j] - x1;
        float dy = bodies.y[j] - y1;
        float dz = bodies.z[j] - z1;

        float r2 = dx * dx + dy * dy + dz * dz + 1e-10f; // Softening factor
        float inv_r3 = rsqrtf(r2 * r2 * r2); // Inverse of r^3

        ax += bodies.Gm[j] * dx * inv_r3;
        ay += bodies.Gm[j] * dy * inv_r3;
        az += bodies.Gm[j] * dz * inv_r3;
    }

    bodies.ax[i] = ax;
    bodies.ay[i] = ay;
    bodies.az[i] = az;
}

__global__ void step_forward_kernel(int n, Bodies bodies, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    // Update velocities
    bodies.vx[i] += bodies.ax[i] * dt;
    bodies.vy[i] += bodies.ay[i] * dt;
    bodies.vz[i] += bodies.az[i] * dt;

    // Update positions
    bodies.x[i] += bodies.vx[i] * dt;
    bodies.y[i] += bodies.vy[i] * dt;
    bodies.z[i] += bodies.vz[i] * dt;
}

class GravSim {
public:
    bool verbose;
    enum class configuration {CIRCULAR, ELLIPTICAL_2D, CIRCULAR_RANDOM, ELLIPTICAL_3D};
private:
    ofstream graphfile;
    vector<string> names;
    unordered_map<string, uint32_t> orbit_map;
    vector<struct body> bodies;

    Bodies device_bodies;
    int n; // # of bodies
    float dt;
    uint64_t num_steps;
    uint64_t timestep;

    void read_line(ifstream &infile, configuration config);
    void add_body(const string& name, uint32_t orbiting_body, float m, float x, float y, float z, float vx, float vy, float vz);
    void add_body_circular(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod);
    void add_body_circular_random(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod);
    void add_body_elliptical(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod);
public:
    GravSim(const char filename[], float timestep_dt, float duration, bool verbose_flag, uint32_t print_every, uint32_t graph_every, configuration config);
    ~GravSim();
    GravSim(const GravSim &orig) = delete;
    GravSim& operator=(const GravSim &rhs) = delete;
    void compute_acceleration_cuda(int threads_per_block = 256);
    void step_forward_cuda(float dt, int threads_per_block = 256);
    void print_system() const;
    void graph_system();
};

// GravSim Methods

void GravSim::add_body(const string& name, uint32_t orbiting_body, float m, float x, float y, float z, float vx, float vy, float vz) {
    bodies.push_back({uint32_t(names.size()), G * m, x, y, z, vx, vy, vz, 0, 0, 0});
    names.push_back(name);
    orbit_map[name] = names.size() - 1;
}

void GravSim::add_body_circular(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod) {
    float Gm = bodies[orbiting_body].Gm;
    float v0 = sqrt(Gm / a); // Orbit velocity
    if (orbPeriod < 0) v0 = -v0;
    float x = bodies[orbiting_body].x + a;
    add_body(name, orbiting_body, m, x, 0, 0, 0, v0, 0);
}

void GravSim::add_body_circular_random(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod) {
    float v0 = sqrt(bodies[orbiting_body].Gm / a);
    if (orbPeriod < 0) v0 = -v0;
    float angle = dis(gen) * 2 * PI;
    float x = bodies[orbiting_body].x + a * cos(angle);
    float y = bodies[orbiting_body].y + a * sin(angle);
    add_body(name, orbiting_body, m, x, y, 0, -v0 * sin(angle), v0 * cos(angle), 0);
}

void GravSim::add_body_elliptical(const string& name, uint32_t orbiting_body, float m, float a, float e, float orbPeriod) {
    float Gm = bodies[orbiting_body].Gm;
    float v0 = sqrt(Gm * (1 - e * e) / a); // Vis-viva equation
    float angle = dis(gen) * 2 * PI;
    add_body(name, orbiting_body, m, a * cos(angle), a * sin(angle), 0, -v0 * sin(angle), v0 * cos(angle), 0);
}

void GravSim::read_line(ifstream &infile, configuration config) {
    char buffer[4096];
    infile.getline(buffer, 4096);
    if (infile.fail()) return;
    if (buffer[0] == '#') return; //Omit comments 
    if (buffer[0] == ' ' || buffer[0] == '\0') return; // Omit blank lines
    stringstream ss(buffer);
    string name, orbits;
    float mass, diam, perihelion, aphelion, orbPeriod, rotationalPeriod, axialtilt, orbinclin;
    ss >> name >> orbits >> mass >> diam >> perihelion >> aphelion >> orbPeriod >> rotationalPeriod >> axialtilt >> orbinclin;
    if (bodies.size() == 0) {
        // Sun first
        add_body(name, 0, mass, 0, 0, 0, 0, 0, 0);
        return;
    }
    // Use orbits to find the parent body
    auto it = orbit_map.find(orbits);
    uint32_t orbiting_body = (it != orbit_map.end()) ? it->second : 0; // Default to Sun if not found

    if (config == configuration::CIRCULAR) {    
        add_body_circular(name, orbiting_body, mass, perihelion, 0, orbPeriod);
    } else if (config == configuration::CIRCULAR_RANDOM) {
        add_body_circular_random(name, orbiting_body, mass, perihelion, 0, orbPeriod);
    } else if (config == configuration::ELLIPTICAL_2D) {
        add_body_elliptical(name, orbiting_body, mass, perihelion, aphelion, orbPeriod);
    } else if (config == configuration::ELLIPTICAL_3D) {
        // TODO: Implement elliptical 3D
    }
}

GravSim::GravSim(const char filename[], float timestep_dt, float duration, bool verbose_flag, uint32_t print_every_param, uint32_t graph_every_param, configuration config) 
    : verbose(verbose_flag), graphfile("solargraph.dat"), dt(timestep_dt), num_steps(duration / timestep_dt), timestep(0) {

    ifstream infile(filename);
    if (!infile.is_open()) {
        cerr << "Failed to open input file: " << filename << endl;
        exit(EXIT_FAILURE);
    }
    while (infile) {
        read_line(infile, config);
    }
    infile.close();

    n = bodies.size();

    // Allocate Unified Memory https://www.olcf.ornl.gov/wp-content/uploads/2019/06/06_Managed_Memory.pdf
    hipError_t err;
    err = hipMallocManaged(&device_bodies.Gm, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (Gm): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.x, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (x): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.y, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (y): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.z, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (z): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.vx, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (vx): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.vy, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (vy): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.vz, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (vz): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.ax, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (ax): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.ay, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (ay): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.az, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (az): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.old_ax, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (old_ax): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.old_ay, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (old_ay): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }
    err = hipMallocManaged(&device_bodies.old_az, n * sizeof(float));
    if (err != hipSuccess) { cerr << "CUDA malloc error (old_az): " << hipGetErrorString(err) << endl; exit(EXIT_FAILURE); }

    for (int i = 0; i < n; i++) {
        device_bodies.Gm[i] = bodies[i].Gm;
        device_bodies.x[i] = bodies[i].x;
        device_bodies.y[i] = bodies[i].y;
        device_bodies.z[i] = bodies[i].z;
        device_bodies.vx[i] = bodies[i].vx;
        device_bodies.vy[i] = bodies[i].vy;
        device_bodies.vz[i] = bodies[i].vz;
        device_bodies.ax[i] = bodies[i].ax;
        device_bodies.ay[i] = bodies[i].ay;
        device_bodies.az[i] = bodies[i].az;
        device_bodies.old_ax[i] = bodies[i].old_ax;
        device_bodies.old_ay[i] = bodies[i].old_ay;
        device_bodies.old_az[i] = bodies[i].old_az;
    }

    // Main simulation loop
    cout << "Starting simulation with " << n << " bodies, num_steps=" << num_steps << endl;
    for (int i = 0; i < num_steps; i++) {
        // Old accelerations
        for (int j = 0; j < n; j++) {
            device_bodies.old_ax[j] = device_bodies.ax[j];
            device_bodies.old_ay[j] = device_bodies.ay[j];
            device_bodies.old_az[j] = device_bodies.az[j];
        }

        compute_acceleration_cuda();

        step_forward_cuda(dt);

        if (verbose) {
            timestep = i;
            if (i % print_every == 0) {
                print_system();
            }
            if (i % graph_every == 0) {
                graph_system();
            }
        }
    }

    hipDeviceSynchronize();
}

GravSim::~GravSim() {
    // Free Unified Memory
    hipFree(device_bodies.Gm);
    hipFree(device_bodies.x);
    hipFree(device_bodies.y);
    hipFree(device_bodies.z);
    hipFree(device_bodies.vx);
    hipFree(device_bodies.vy);
    hipFree(device_bodies.vz);
    hipFree(device_bodies.ax);
    hipFree(device_bodies.ay);
    hipFree(device_bodies.az);
    hipFree(device_bodies.old_ax);
    hipFree(device_bodies.old_ay);
    hipFree(device_bodies.old_az);
}

void GravSim::compute_acceleration_cuda(int threads_per_block) {
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    compute_acceleration_kernel<<<blocks, threads_per_block>>>(n, device_bodies);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Failed to launch compute_acceleration_kernel: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
}

void GravSim::step_forward_cuda(float dt, int threads_per_block) {
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    step_forward_kernel<<<blocks, threads_per_block>>>(n, device_bodies, dt);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Failed to launch step_forward_kernel: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
}

void GravSim::print_system() const {
    for (int i = 0; i < n; i++) {
        cout << names[i] << " " 
             << device_bodies.x[i] << "," 
             << device_bodies.y[i] << "," 
             << device_bodies.z[i] << "   " 
             << device_bodies.vx[i] << "," 
             << device_bodies.vy[i] << "," 
             << device_bodies.vz[i] << endl;
    }
}

void GravSim::graph_system() {
    for (int i = 0; i < n; i++) {
        graphfile << names[i] << ' ' 
                  << device_bodies.x[i] << ' ' 
                  << device_bodies.y[i] << ' ' 
                  << device_bodies.z[i] << ' ';
    }
    graphfile << '\n';
}

// Main 

int main(int argc, char **argv) {
    const char *filename = (argc > 1) ? argv[1] : "solarsys.dat";
    float dt = 1000.0f; // Timestep in seconds
    float duration = year; // One year
    bool verbose = true;
    uint32_t print_every = static_cast<uint32_t>(31536000 / dt); // Print once per year
    uint32_t graph_every = static_cast<uint32_t>(86400 / dt); // Graph once per day

    GravSim sim(filename, dt, duration, verbose, print_every, graph_every, GravSim::configuration::CIRCULAR_RANDOM);
    sim.print_system();
    sim.graph_system();

    return 0;
}

